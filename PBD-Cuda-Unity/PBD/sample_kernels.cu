#include "hip/hip_runtime.h"
﻿#include "sample_kernels.cuh"

__global__ void writeVertexBuffer(float4 *pos, int size, float time)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

    // write output vertex
    if (x < size)
    {
        pos[x] = make_float4(cos(2 * HIP_PI_F * time / x),
                             sin(2 * HIP_PI_F * time / x), 0.0f, 1.0f);
    }
}

void kernelCallerWriteBuffer(const dim3 dimGrid, const dim3 dimBlock,
                             float4 *vertexPtr, const int size,
                             const float time)
{
    writeVertexBuffer<<<dimGrid, dimBlock>>>(vertexPtr, size, time);
}